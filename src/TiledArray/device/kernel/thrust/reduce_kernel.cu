#include "hip/hip_runtime.h"
/*
 *  This file is a part of TiledArray.
 *  Copyright (C) 2018  Virginia Tech
 *
 *  This program is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 3 of the License, or
 *  (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
 *  Chong Peng
 *  Department of Chemistry, Virginia Tech
 *  May 8, 2019
 *
 */

#include <TiledArray/device/kernel/reduce_kernel.h>
#include <TiledArray/device/kernel/thrust/reduce_kernel.h>

namespace TiledArray::device {

// foreach(i) result *= arg[i]
int product_kernel(const int *arg, std::size_t n, const Stream& stream){
  return product_reduce_kernel_thrust(arg, n, stream);
}

float product_kernel(const float *arg, std::size_t n, const Stream& stream){
  return product_reduce_kernel_thrust(arg, n, stream);
}

double product_kernel(const double *arg, std::size_t n, const Stream& stream){

  return product_reduce_kernel_thrust(arg, n, stream);
}

std::complex<float> product_kernel(const std::complex<float> *arg, std::size_t n, const Stream& stream){
  return product_reduce_kernel_thrust(arg, n, stream);
}

std::complex<double> product_kernel(const std::complex<double> *arg, std::size_t n, const Stream& stream){

  return product_reduce_kernel_thrust(arg, n, stream);
}

// foreach(i) result += arg[i]
int sum_kernel(const int *arg, std::size_t n, const Stream& stream){
  return sum_reduce_kernel_thrust(arg, n, stream);
}

float sum_kernel(const float *arg, std::size_t n, const Stream& stream){
  return sum_reduce_kernel_thrust(arg, n, stream);
}

double sum_kernel(const double *arg, std::size_t n, const Stream& stream){
  return sum_reduce_kernel_thrust(arg, n, stream);
}

std::complex<float> sum_kernel(const std::complex<float> *arg, std::size_t n, const Stream& stream){
  return sum_reduce_kernel_thrust(arg, n, stream);
}

std::complex<double> sum_kernel(const std::complex<double> *arg, std::size_t n, const Stream& stream){
  return sum_reduce_kernel_thrust(arg, n, stream);
}

// foreach(i) result = max(result, arg[i])
int max_kernel(const int *arg, std::size_t n, const Stream& stream){
  return max_reduce_kernel_thrust(arg, n, stream);
}

float max_kernel(const float *arg, std::size_t n, const Stream& stream){
  return max_reduce_kernel_thrust(arg, n, stream);
}

double max_kernel(const double *arg, std::size_t n, const Stream& stream){
  return max_reduce_kernel_thrust(arg, n, stream);
}

// foreach(i) result = min(result, arg[i])
int min_kernel(const int *arg, std::size_t n, const Stream& stream){
  return min_reduce_kernel_thrust(arg, n, stream);
}

float min_kernel(const float *arg, std::size_t n, const Stream& stream){
  return min_reduce_kernel_thrust(arg, n, stream);
}

double min_kernel(const double *arg, std::size_t n, const Stream& stream){
  return min_reduce_kernel_thrust(arg, n, stream);
}

// foreach(i) result = max(result, abs(arg[i]))
int absmax_kernel(const int *arg, std::size_t n, const Stream& stream){
  return absmax_reduce_kernel_thrust(arg, n, stream);
}

float absmax_kernel(const float *arg, std::size_t n, const Stream& stream){
  return absmax_reduce_kernel_thrust(arg, n, stream);
}

double absmax_kernel(const double *arg, std::size_t n, const Stream& stream){
  return absmax_reduce_kernel_thrust(arg, n, stream);
}

std::complex<float> absmax_kernel(const std::complex<float> *arg, std::size_t n, const Stream& stream){
  return absmax_reduce_kernel_thrust(arg, n, stream);
}

std::complex<double> absmax_kernel(const std::complex<double> *arg, std::size_t n, const Stream& stream){
  return absmax_reduce_kernel_thrust(arg, n, stream);
}

// foreach(i) result = min(result, abs(arg[i]))
int absmin_kernel(const int *arg, std::size_t n, const Stream& stream){
  return absmin_reduce_kernel_thrust(arg, n, stream);
}

float absmin_kernel(const float *arg, std::size_t n, const Stream& stream){
  return absmin_reduce_kernel_thrust(arg, n, stream);
}

double absmin_kernel(const double *arg, std::size_t n, const Stream& stream){
  return absmin_reduce_kernel_thrust(arg, n, stream);
}

std::complex<float> absmin_kernel(const std::complex<float> *arg, std::size_t n, const Stream& stream){
  return absmin_reduce_kernel_thrust(arg, n, stream);
}

std::complex<double> absmin_kernel(const std::complex<double> *arg, std::size_t n, const Stream& stream){
  return absmin_reduce_kernel_thrust(arg, n, stream);
}

}  // namespace TiledArray::device
